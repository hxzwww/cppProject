#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

__global__ void cudaMerge(int* input, int* output, int start, int mid, int end) {
    int i = start + blockIdx.x * blockDim.x + threadIdx.x;

    if (i < end) {
        int a = (i < mid) ? input[i] : INT_MAX;
        int b = (i >= mid && i < end) ? input[i] : INT_MAX;

        output[i] = (a < b) ? a : b;
    }
}

void cudaMergeSort(int* d_data, int* d_temp, int start, int end, int N) {
    if (end - start <= 1) return;

    int mid = (start + end) / 2;

    cudaMergeSort(d_data, d_temp, start, mid, N);
    cudaMergeSort(d_data, d_temp, mid, end, N);

    int threadsPerBlock = 256;
    int blocksPerGrid = (end - start + threadsPerBlock - 1) / threadsPerBlock;
    cudaMerge<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_temp, start, mid, end);

    hipMemcpy(d_data + start, d_temp + start, (end - start) * sizeof(int), hipMemcpyDeviceToDevice);
}


void mergeSort(std::vector<int>& data) {
    int N = data.size();

    int* d_data;
    int* d_temp;
    hipMalloc((void**)&d_data, N * sizeof(int));
    hipMalloc((void**)&d_temp, N * sizeof(int));

    hipMemcpy(d_data, data.data(), N * sizeof(int), hipMemcpyHostToDevice);

    cudaMergeSort(d_data, d_temp, 0, N, N);

    hipMemcpy(data.data(), d_data, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipFree(d_temp);

}
